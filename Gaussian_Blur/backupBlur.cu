#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <stdlib.h>

#define cimg_use_jpeg
#include "CImg.h"

#define THREADS 128

static void CudaTest(const char *msg)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}


static __global__ void blur(unsigned char *d_rout, unsigned char *d_gout, unsigned char *d_bout, unsigned char *d_rO, unsigned char *d_gO, unsigned char *d_bO, const int H, const int W) {

  int index= (threadIdx.x) + blockIdx.x * blockDim.x;
  int r,c; 
  //compute the blur
  if (index > 0 && index < (H) * (W)) {
    r = index / W;
    c = index % W;

    //red
    d_rout[index] = (d_rO[(r+1) * W + c] + d_rO[(r-1) * W + c] +
                           d_rO[r * W + (c+1)] + d_rO[(r+1) * W + (c+1)] + d_rO[(r-1) * W + (c+1)] +
                           d_rO[r * W + (c-1)] + d_rO[(r+1) * W + (c-1)] + d_rO[(r-1) * W + (c-1)]) / 8;

    //green
    d_gout[index] = (d_gO[(r+1) * W + c] + d_gO[(r-1) * W + c] +
                           d_gO[r * W + (c+1)] + d_gO[(r+1) * W + (c+1)] + d_gO[(r-1) * W + (c+1)] +
                           d_gO[r * W + (c-1)] + d_gO[(r+1) * W + (c-1)] + d_gO[(r-1) * W + (c-1)]) / 8;

    //blue
    d_bout[index] = (d_bO[(r+1) * W + c] + d_bO[(r-1) * W + c] +
                           d_bO[r * W +(c+1)] + d_bO[(r+1) * W + (c+1)] + d_bO[(r-1) * W + (c+1)] +
                           d_bO[r * W + (c-1)] + d_bO[(r+1) * W + (c-1)] + d_bO[(r-1) * W + (c-1)]) / 8;
  }
}

int main(int argc, char *argv[]) {
  struct timeval start, end;
  double runtime = 0.0, total = 0.0;

  if (argc != 3) {
    fprintf(stderr, "usage: exe, input file, number of groups\n"); exit(-1);
  }
  
  // import image from jpg file
  cimg_library::CImg<unsigned char> input_img(argv[1]);

  const int NUM_PART = atoi(argv[2]);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("Running on %s\n", prop.name);

  //create Height/Width variables for readability
  const int H = input_img.height();
  const int W = input_img.width();
  const int Hout = H + 2;
  const int Wout = W + 2;

  //create height/offset for partitions
  int offsetH = H / NUM_PART; //height for each group, not padded

  //create GPU variables
  unsigned char *d_rO, *d_gO, *d_bO, *d_rout, *d_gout, *d_bout;

  //allocate CPU arrays
  unsigned char* rO = (unsigned char*)calloc(Wout * Hout, sizeof(unsigned char));
  unsigned char* gO = (unsigned char*)calloc(Wout * Hout, sizeof(unsigned char));
  unsigned char* bO = (unsigned char*)calloc(Wout * Hout, sizeof(unsigned char));
  unsigned char* rout = (unsigned char*)malloc(W * H * sizeof(unsigned char));
  unsigned char* gout = (unsigned char*)malloc(W * H * sizeof(unsigned char));
  unsigned char* bout = (unsigned char*)malloc(W * H * sizeof(unsigned char)); 

  //allocate GPU memory for arrays
  hipMalloc((void**)&d_rO, Wout * (offsetH+2) * sizeof(unsigned char));
  hipMalloc((void**)&d_gO, Wout * (offsetH+2) * sizeof(unsigned char));
  hipMalloc((void**)&d_bO, Wout * (offsetH+2) * sizeof(unsigned char));
  hipMalloc((void**)&d_rout, Wout * (offsetH+2) * sizeof(unsigned char));
  hipMalloc((void**)&d_gout, Wout * (offsetH+2) * sizeof(unsigned char));
  hipMalloc((void**)&d_bout, Wout * (offsetH+2) * sizeof(unsigned char));

  //pad the image
  for(int c = 0; c< W; c++) {
    for(int r = 0; r < H; r++) {
      rO[(r+1) * W + (c+1) ] = input_img(c, r, 0);
      gO[(r+1) * W + (c+1) ] = input_img(c, r, 1);
      bO[(r+1) * W + (c+1) ] = input_img(c, r, 2);
    }
  }

  //create new image   
  cimg_library::CImg<unsigned char> output_img(W, H, 1, 3);

  //loop over number of groups, calculate portion of blur for each
  for(int i = 0; i < NUM_PART; i++) {
    //send over padded image info to GPU
    if(hipSuccess != hipMemcpy(d_rO, &rO[(i * (offsetH)) * Wout], Wout * (offsetH+2) * sizeof(unsigned char), hipMemcpyHostToDevice)) fprintf(stderr, "copy to device failed\n");
    if(hipSuccess != hipMemcpy(d_gO, &gO[(i * (offsetH)) * Wout], Wout * (offsetH+2) * sizeof(unsigned char), hipMemcpyHostToDevice)) fprintf(stderr, "copy to device failed\n");
    if(hipSuccess != hipMemcpy(d_bO, &bO[(i * (offsetH)) * Wout], Wout * (offsetH+2) * sizeof(unsigned char), hipMemcpyHostToDevice)) fprintf(stderr, "copy to device failed\n");

    //launch kernel
    gettimeofday(&start, NULL);
    blur<<<((offsetH+1) * (W+1) + THREADS - 1) / THREADS, THREADS>>>(d_rout, d_gout, d_bout, d_rO, d_gO, d_bO, (offsetH+1), (W+1));
    gettimeofday(&end, NULL);

    runtime = end.tv_sec + (end.tv_usec / 1000000.0) - start.tv_sec - (start.tv_usec / 1000000.0);
    total+= runtime;

    //send the blurred image info back to CPU
    if(hipSuccess != hipMemcpy(&rout[((i * (offsetH)) * Wout)], d_rout, W * (offsetH) * sizeof(unsigned char), hipMemcpyDeviceToHost)) fprintf(stderr, "copy to host failed\n");
    if(hipSuccess != hipMemcpy(&gout[((i * (offsetH)) * Wout)], d_gout, W * (offsetH) * sizeof(unsigned char), hipMemcpyDeviceToHost)) fprintf(stderr, "copy to host failed\n");
    if(hipSuccess != hipMemcpy(&bout[((i * (offsetH)) * Wout)], d_bout, W * (offsetH) * sizeof(unsigned char), hipMemcpyDeviceToHost)) fprintf(stderr, "copy to host failed\n");
  }

  printf("\nCompute time for Blur: %.8f s\n", runtime);

  for(int c = 0; c < W; c++) {
    for(int r = 0; r < H; r++) {
      output_img(c, r, 0) = rout[(r)* W + (c)];
      output_img(c, r, 1) = gout[(r)* W + (c)];
      output_img(c, r, 2) = bout[(r)* W + (c)];
    }
  }
 
  //save output to file
  output_img.save_jpeg("output.jpg");

  hipFree(d_rO); hipFree(d_gO); hipFree(d_bO);
  hipFree(d_rout); hipFree(d_gout); hipFree(d_bout);
  free(rO); free(gO); free(bO); free(rout); free(gout); free(bout);
  return 0;
}
