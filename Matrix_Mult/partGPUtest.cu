#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

const int THREADS = 512;

static void CudaTest(const char *msg)
{
  hipError_t e;
  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}

__global__ void matrixMult(const int N, int part, double *d_A, double *d_B, double *d_C) 
{
  int index = threadIdx.x + blockIdx.x * blockDim.x; 
  double pSum = 0.0;

  if (index < part * N) 
  { 
    int r = index / N;
    int c = index % N;

    for (int i = 0; i < N; i++) {
      pSum += d_A[r * N + i] * d_B[i * N + c];
    }
    d_C[index] = pSum; 
  } 
}

int main(int argc, char* argv[])
{
  if(argc != 3) {fprintf(stderr, "usage: <exe>, size_of_array, num_groups\n"); exit(-1);}

  const int N = atoi(argv[1]);
  double *d_A, *d_B, *d_C; //device variables 

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  printf("Running on %s\n", prop.name);

  //timing
  struct timeval start, end; 
  double runtime = 0.0, total = 0.0;

  const int factor = atoi(argv[2]);
  int part = N / factor;

  //allocate on CPU
  double *arrayA = (double *)malloc(N * N * sizeof(double));
  double *arrayB = (double *)malloc(N * N * sizeof(double));
  double *arrayC = (double *)malloc(N * N * sizeof(double)); 

  //fill array inputs on CPU
  for (int i = 0; i < N; i++)
    for (int j = 0; j < N; j++) {     
      arrayA[i * N + j] = (i * N + j) + rand() % 100000 / 100000.0;
      arrayB[i * N + j] = 2 * rand() % 100000 / 100000.0;
    } 

  //allocate space on GPU
  hipMalloc((void**) &d_A, part * N * sizeof(double)); CudaTest("failed allocation");
  hipMalloc((void**) &d_B, N * N * sizeof(double)); CudaTest("failed allocation");
  hipMalloc((void**) &d_C, part * N * sizeof(double)); CudaTest("failed allocation");

  hipMemcpy(d_B, &arrayB, N * N * sizeof(double), hipMemcpyHostToDevice); CudaTest("failed to send data to GPU");

  for(int i = 0; i < factor; i++) {
    //send part data to GPU for MM
    hipMemcpy(d_A, &arrayA[(i * part) * N], part * N * sizeof(double), hipMemcpyHostToDevice); CudaTest("failed to send data to GPU");
 
    //run first kernel
    gettimeofday(&start, NULL);   
    matrixMult<<<((part * N + THREADS -1)/THREADS), THREADS>>> (N, part, d_A, d_B, d_C); CudaTest("failed kernel"); 
    gettimeofday(&end, NULL);
    runtime = end.tv_sec + (end.tv_usec / 1000000.0) - start.tv_sec - (start.tv_usec / 1000000.0);
    total += runtime;
   
    //send part data back to CPU for MM
    hipMemcpy(&arrayC[(i * part) * N], d_C, part * N * sizeof(double), hipMemcpyDeviceToHost); CudaTest("failed to send data back");
  }
  printf("\nCompute time for Matrix Multiply: %.4f s\n", total);
 
/*
  //check result
  for (int i = 0; i < N; i++) 
    for(int j = 0; j < N; j++) 
    { 
      printf("Array C: %.2lf \n", arrayC[i * N + j]);
      printf("Array B: %.2lf \n", arrayB[i * N + j]);
    }
*/

  //free memory
  hipFree(d_A); hipFree(d_B); hipFree(d_C);
  free(arrayA); free(arrayB); free(arrayC);
  return 0;
}
